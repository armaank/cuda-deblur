
#include <hip/hip_runtime.h>
/* Benchmarking algorithm performance via GPU time */

/* based off example here: https://devblogs.nvidia.com/how-implement-performance-metrics-cuda-cc */

struct GpuTimer
{
	hipEvent_t start_val, stop_val;

	GpuTimer()
	{
		hipEventCreate(&start_val);
		hipEventCreate(&stop_val);
	}

	~GpuTimer()
	{
		hipEventDestroy(start_val);
		hipEventDestroy(stop_val);
	}

	void start()
	{
		hipEventRecord(start_val, 0);
	}

	void stop()
	{
		hipEventRecord(stop_val, 0);
	}

	float elapsed_time()
	{
		float elapsed;
		hipEventSynchronize(stop_val);
		hipEventElapsedTime(&elapsed, start_val, stop_val);
		return elapsed;
	}
};

