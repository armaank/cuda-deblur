#include "hip/hip_runtime.h"


#include <vector>
#include <iostream>

#include "gpuLucyRichardson.cu"
#include "../benchmarks/metrics.hpp" 
#include "../benchmarks/gputime.cu"
#include "pngConnector.hpp"
// #include "cpu/ops.hpp" for later, so we don't need to re-write createMatrix, gaussian, sharpen here

#define NUM_ITERATIONS 1


void runLucyRichardson(double *filter_ptr, double *filter_mirror_ptr, double *image_ptr, double *output_ptr, 
    const Image &target_image, const std::string &output_file, int filter_width, int filter_height);
void runSimpleFilter(const Matrix &kernel, const Image &blurry_image, const Image &target_image, const std::string &output_file);
void gpuDeblur(double *filter_ptr, double *filter_mirror_ptr, double *image_ptr, double *output_ptr,
    const Image &target_image, const std::string &output_file, int filter_width, int filter_height, double *s_filter_ptr, int s_filter_width, int s_filter_height);


Matrix createMatrix(const int height, const int width);
Matrix gaussian(const int height, const int width, const double sigma);
Matrix sharpen(const int height, const int width);

double *image2ptr(const Image &input);
double *matrix2ptr(const Matrix &input);
Image ptr2image(const double *input, const int width, const int height);


int main(int argc, char **argv)
{
    if (argc <= 3)
    {
        std::cerr << "error: specify input and output files" << std::endl;
        return -1;
    }
    std::string input_file = argv[1];  // blurry image
    std::string output_file = argv[2]; // deblurred image
    std::string target_file = argv[3]; // target image 'ground truth';


    std::cout << "Loading image from" << input_file << std::endl;
    Image image = loadImage(input_file);
    Image target_image = loadImage(target_file);

    double *image_ptr  = image2ptr(image);
    double *output_ptr = new (std::nothrow) double[3*image[0].size()*image[0][0].size()];



    /////////////////////////////////////////////////////////////////////////

    // Kernel: gaussian 3x3
    Matrix filter = gaussian(3, 3, 1);
    int filter_width  = filter[0].size(); 
    int filter_height = filter.size(); 
    Matrix filter_m = createMatrix(filter_height, filter_width);
    for (int i = 0; i < filter_height; i++)
        for (int j = 0; j < filter_width; j++)
            filter_m[i][j] = filter[j][i];
    
    Matrix s_filter = sharpen(3,3);
    int s_filter_width = s_filter[0].size();
    int s_filter_height = s_filter.size();
    

    double *filter_ptr = matrix2ptr(filter);    
    double *filter_mirror_ptr = matrix2ptr(filter_m);
    double *s_filter_ptr = matrix2ptr(s_filter);

//    runLucyRichardson(filter_ptr, filter_mirror_ptr, image_ptr, output_ptr, target_image, 
  //      output_file+"_gaussKernel3"+ ".png", filter_width, filter_height);
    gpuDeblur(filter_ptr, filter_mirror_ptr, image_ptr, output_ptr, target_image, output_file + "_gaussKernel3"+".png", filter_width, filter_height, s_filter_ptr, s_filter_width, s_filter_height);



    // Kernel: gaussian 7x7
    // filter = gaussian(7, 7, 1);
    // filter_width  = filter[0].size(); 
    // filter_height = filter.size(); 
    // filter_m = createMatrix(filter_height, filter_width);
    // for (int i = 0; i < filter_height; i++)
    //     for (int j = 0; j < filter_width; j++)
    //         filter_m[i][j] = filter[j][i];

    // filter_ptr = matrix2ptr(filter);    
    // filter_mirror_ptr = matrix2ptr(filter_m);
    // runLucyRichardson(filter_ptr, filter_mirror_ptr, image_ptr, output_ptr, target_image, 
    //     output_file+"_gaussKernel7"+ ".png", filter_width, filter_height);

    // filter = sharpen(3,3);
    // runSimpleFilter(filter, image, target_image, output_file+"_sharpen3"+".png");

    std::cout << "Done!" << std::endl;
    
    /////////////////////////////////////////////////////////////////////////


    return 0;
}

// void runSimpleFilter(const Matrix &filter, const Image &blurry_image, const Image &target_image, const std::string &output_file)
// {
//     std::cout << "running lucy iterations..." << std::endl;
    
//     /* initalize gpu timers */
//     GpuTimer gputime_gpu;
    
//     int element_count = 3*blurry_image[0].size()*blurry_image[0][0].size(); 
//     int size = element_count*sizeof(double);
//     double *image_ptr  = image2ptr(blurry_image);
//     double *output_ptr = new (std::nothrow) double[element_count];
//     double *filter_ptr = matrix2ptr(filter);    

//     hipError_t err = hipSuccess;  // Error code to check return values for CUDA calls

//     gputime_gpu.start();

//     // Allocate the device input vector f
//     double *d_f = NULL;
//     err = hipMalloc((void **)&d_f, size);
//     if (err != hipSuccess)
//     {
//         fprintf(stderr, "Failed to allocate device vector f (error code %s)!\n", hipGetErrorString(err));
//         exit(EXIT_FAILURE);
//     }

//     // Allocate the device input vector g
//     double *d_g = NULL;
//     err = hipMalloc((void **)&d_g, size);
//     if (err != hipSuccess)
//     {
//         fprintf(stderr, "Failed to allocate device vector g (error code %s)!\n", hipGetErrorString(err));
//         exit(EXIT_FAILURE);
//     }

//     // Allocate the device output vector c
//     double *d_c = NULL;
//     err = hipMalloc((void **)&d_c, size);
//     if (err != hipSuccess)
//     {
//         fprintf(stderr, "Failed to allocate device vector c (error code %s)!\n", hipGetErrorString(err));
//         exit(EXIT_FAILURE);
//     }

//     // Copy the host input vectors f, g, and c in host memory to the device input vectors in device memory
//     printf("Copy input data from the host memory to the CUDA device\n");
//     err = hipMemcpy(d_g, filter_ptr, filter.size()*filter[0].size(), hipMemcpyHostToDevice);
//     if (err != hipSuccess)
//     {
//         fprintf(stderr, "Failed to copy vector g from host to device (error code %s)!\n", hipGetErrorString(err));
//         exit(EXIT_FAILURE);
//     }

//     err = hipMemcpy(d_c, image_ptr, size, hipMemcpyHostToDevice);
//     if (err != hipSuccess)
//     {
//         fprintf(stderr, "Failed to copy vector c from host to device (error code %s)!\n", hipGetErrorString(err));
//         exit(EXIT_FAILURE);
//     }

//     // convolve filter and image
//     int threadsPerBlock = 256;
//     int blocksPerGrid =(blurry_image[0].size()*blurry_image[0][0].size() + threadsPerBlock - 1) / threadsPerBlock;
//     kernel_convolve<<<blocksPerGrid, threadsPerBlock>>>(d_c, d_g, d_f, blurry_image[0][0].size(), blurry_image[0].size());
//     err = hipGetLastError();
//     if (err != hipSuccess)
//     {
//         fprintf(stderr, "Failed to launch KernelConvolve kernel (error code %s)!\n", hipGetErrorString(err));
//         exit(EXIT_FAILURE);
//     }

//     // de-allocate device arrays
//     err = hipFree(d_f);
//     if (err != hipSuccess)
//     {
//         fprintf(stderr, "Failed to free device vector f (error code %s)!\n", hipGetErrorString(err));
//         exit(EXIT_FAILURE);
//     }

//     err = hipFree(d_c);
//     if (err != hipSuccess)
//     {
//         fprintf(stderr, "Failed to free device vector c (error code %s)!\n", hipGetErrorString(err));
//         exit(EXIT_FAILURE);
//     }

//     err = hipFree(d_g);
//     if (err != hipSuccess)
//     {
//         fprintf(stderr, "Failed to free device vector g (error code %s)!\n", hipGetErrorString(err));
//         exit(EXIT_FAILURE);
//     }

//     // compute psnr
//     gputime_gpu.stop();

//     Image output = ptr2image(output_ptr, blurry_image[0][0].size(), blurry_image[0].size());

//     std::cout << "Total time Elapsed - GPU: " << gputime_gpu.elapsed_time() << " ms" << std::endl;

//     std::cout << "PSNR: " << psnr(output, target_image) << std::endl;

//     saveImage(output, output_file);
//     std::cout << "Image saved to: " << output_file << std::endl;
// }

void gpuDeblur(double *filter_ptr, double *filter_mirror_ptr, double *image_ptr, double *output_ptr,
    const Image &target_image, const std::string &output_file, int filter_width, int filter_height, double *s_filter_ptr, int s_filter_width, int s_filter_height)
{
    /* initalize gpu timers */
    GpuTimer gputime_gpu;
    gputime_gpu.start();
    int height = target_image[0].size();
    int width = target_image[0][0].size();
    int filter_size = filter_width*filter_height*sizeof(double);
    int element_count = 3*height*width;
    int size = element_count*sizeof(double);
    int s_filter_size = s_filter_width*s_filter_height*sizeof(double);

    hipError_t err = hipSuccess;  // Error code to check return values for CUDA calls

    // Allocate the device output vector f
    double *d_f = NULL;
    err = hipMalloc((void **)&d_f, size);
    if (err != hipSuccess)
    {
        std::cerr << "Failed to allocate device vector f (error code " << hipGetErrorString(err) << ")!" << std::endl;
        exit(EXIT_FAILURE);
    }

    // Allocate the device input vector g
    double *d_g = NULL;
    err = hipMalloc((void **)&d_g, size);
    if (err != hipSuccess)
    {
        std::cerr << "Failed to allocate device vector g (error code " << hipGetErrorString(err) << ")!" << std::endl;
        exit(EXIT_FAILURE);
    }

	
    // Allocate the device input vector g_m
    double *d_g_m = NULL;
    err = hipMalloc((void **)&d_g_m, size);
    if (err != hipSuccess)
    {
        std::cerr << "Failed to allocate device vector g_m (error code " << hipGetErrorString(err) << ")!" << std::endl;
        exit(EXIT_FAILURE);
    }

    // Allocate device inout vector s (sharpening filter)
    double *d_s = NULL;
    err = hipMalloc((void **)&d_s, size);
    if (err != hipSuccess)
    {
        std::cerr << "Failed to allocate device vector c (error code " << hipGetErrorString(err) << ")!" << std::endl;
        exit(EXIT_FAILURE);
    }

    
    // Allocate the device output vector c
    double *d_c = NULL;
    err = hipMalloc((void **)&d_c, size);
    if (err != hipSuccess)
    {
        std::cerr << "Failed to allocate device vector c (error code " << hipGetErrorString(err) << ")!" << std::endl;
        exit(EXIT_FAILURE);
    }

    // Copy the host input vectors f, g, and c in host memory to the device input vectors in device memory
    std::cout << "Copy input data from the host memory to the CUDA device." << std::endl;
    err = hipMemcpy(d_g, filter_ptr, filter_size, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        std::cerr << "Failed to copy vector g from host to device (error code " << hipGetErrorString(err) << ")!" << std::endl;
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_g_m, filter_mirror_ptr, filter_size, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        std::cerr << "Failed to copy vector g_m from host to device (error code " << hipGetErrorString(err) << ")!" << std::endl;
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_f, image_ptr, size, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector f from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_s, s_filter_ptr, s_filter_size, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector f from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

	
    err = hipMemcpy(d_c, image_ptr, size, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector c from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
   
    // allocate the temporary gpu memory
    double *d_tmp1 = NULL;
    err = hipMalloc((void **)&d_tmp1, size);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector tmp1 (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    double *d_tmp2 = NULL;
    err = hipMalloc((void **)&d_tmp2, size);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector tmp2 (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    double *d_tmp3 = NULL;
    err = hipMalloc((void **)&d_tmp3, size);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector tmp2 (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    std::cout << "running lucy iterations... ";
    for (int i=0; i<NUM_ITERATIONS; ++i)
    {
        std::cout << i+1 << ", " << std::flush;
        updateUnderlyingImg(d_c, d_g, d_g_m, d_f, d_tmp1, d_tmp2, d_tmp3, width, height, filter_width, filter_height, d_s, s_filter_width, s_filter_height);
        //updateUnderlyingImg_old(d_c, d_g, d_g_m, d_f, d_tmp1, d_tmp2, width, height, filter_width, filter_height);
	
    }
    std::cout << std::endl;
    
    // Copy the device result vector in device memory to the host result vector in host memory.
    std::cout << "Copy output data from the CUDA device to the host memory" << std::endl;
    err = hipMemcpy(output_ptr, d_f, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        std::cerr << "Failed to copy vector f from device to host (error code " << hipGetErrorString(err) << ")!" << std::endl;
        exit(EXIT_FAILURE);
    }

    // Free device global memory
    err = hipFree(d_f);
    if (err != hipSuccess)
    {
        std::cerr << "Failed to free device vector f (error code " << hipGetErrorString(err) << ")!" << std::endl;
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_c);
    if (err != hipSuccess)
    {
        std::cerr << "Failed to free device vector c (error code " << hipGetErrorString(err) << ")!" << std::endl;
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_g);
    if (err != hipSuccess)
    {
        std::cerr << "Failed to free device vector g (error code " << hipGetErrorString(err) << ")!" << std::endl;
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_g_m);
    if (err != hipSuccess)
    {
        std::cerr << "Failed to free device vector g_m (error code " << hipGetErrorString(err) << ")!" << std::endl;
        exit(EXIT_FAILURE);
    }
    
    err = hipFree(d_tmp1);
    if (err != hipSuccess)
    {
        std::cerr << "Failed to free device vector tmp1 (error code " << hipGetErrorString(err) << ")!" << std::endl;
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_tmp2);
    if (err != hipSuccess)
    {
        std::cerr << "Failed to free device vector tmp2 (error code " << hipGetErrorString(err) << ")!" << std::endl;
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_tmp3);
    if (err != hipSuccess)
    {
        std::cerr << "Failed to free device vector tmp3 (error code " << hipGetErrorString(err) << ")!" << std::endl;
        exit(EXIT_FAILURE);
    }


    Image output = ptr2image(output_ptr, width, height);
    gputime_gpu.stop();

    std::cout << "Total time Elapsed - GPU: " << gputime_gpu.elapsed_time() << " ms" << std::endl;

    std::cout << "BaselinePSNR: " << psnr(ptr2image(image_ptr, width, height), target_image) << std::endl;
    std::cout << "PSNR: " << psnr(output, target_image) << std::endl;

    saveImage(output, output_file);
    std::cout << "Image saved to: " << output_file << std::endl;

}



void runLucyRichardson(double *filter_ptr, double *filter_mirror_ptr, double *image_ptr, double *output_ptr, 
    const Image &target_image, const std::string &output_file, int filter_width, int filter_height)
{  
    /* initalize gpu timers */
    GpuTimer gputime_gpu;
    gputime_gpu.start();
    int height = target_image[0].size();
    int width = target_image[0][0].size();
    int filter_size = filter_width*filter_height*sizeof(double);
    int element_count = 3*height*width;
    int size = element_count*sizeof(double);

    hipError_t err = hipSuccess;  // Error code to check return values for CUDA calls

    // Allocate the device output vector f
    double *d_f = NULL;
    err = hipMalloc((void **)&d_f, size);
    if (err != hipSuccess)
    {
        std::cerr << "Failed to allocate device vector f (error code " << hipGetErrorString(err) << ")!" << std::endl;
        exit(EXIT_FAILURE);
    }

    // Allocate the device input vector g
    double *d_g = NULL;
    err = hipMalloc((void **)&d_g, size);
    if (err != hipSuccess)
    {
        std::cerr << "Failed to allocate device vector g (error code " << hipGetErrorString(err) << ")!" << std::endl;
        exit(EXIT_FAILURE);
    }

    // Allocate the device input vector g_m
    double *d_g_m = NULL;
    err = hipMalloc((void **)&d_g_m, size);
    if (err != hipSuccess)
    {
        std::cerr << "Failed to allocate device vector g_m (error code " << hipGetErrorString(err) << ")!" << std::endl;
        exit(EXIT_FAILURE);
    }

    // Allocate the device output vector c
    double *d_c = NULL;
    err = hipMalloc((void **)&d_c, size);
    if (err != hipSuccess)
    {
        std::cerr << "Failed to allocate device vector c (error code " << hipGetErrorString(err) << ")!" << std::endl;
        exit(EXIT_FAILURE);
    }

    // Copy the host input vectors f, g, and c in host memory to the device input vectors in device memory
    std::cout << "Copy input data from the host memory to the CUDA device." << std::endl;
    err = hipMemcpy(d_g, filter_ptr, filter_size, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        std::cerr << "Failed to copy vector g from host to device (error code " << hipGetErrorString(err) << ")!" << std::endl;
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_g_m, filter_mirror_ptr, filter_size, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        std::cerr << "Failed to copy vector g_m from host to device (error code " << hipGetErrorString(err) << ")!" << std::endl;
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_f, image_ptr, size, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector f from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_c, image_ptr, size, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector c from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // allocate the temporary gpu memory
    double *d_tmp1 = NULL;
    err = hipMalloc((void **)&d_tmp1, size);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector tmp1 (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    double *d_tmp2 = NULL;
    err = hipMalloc((void **)&d_tmp2, size);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector tmp2 (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    std::cout << "running lucy iterations... ";
    for (int i=0; i<NUM_ITERATIONS; ++i)
    {
        std::cout << i+1 << ", " << std::flush;
//        updateUnderlyingImg_old(d_c, d_g, d_g_m, d_f, d_tmp1, d_tmp2, width, height, filter_width, filter_height);
    }
    std::cout << std::endl;

    // Copy the device result vector in device memory to the host result vector in host memory.
    std::cout << "Copy output data from the CUDA device to the host memory" << std::endl;
    err = hipMemcpy(output_ptr, d_f, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        std::cerr << "Failed to copy vector f from device to host (error code " << hipGetErrorString(err) << ")!" << std::endl;
        exit(EXIT_FAILURE);
    }

    // Free device global memory
    err = hipFree(d_f);
    if (err != hipSuccess)
    {
        std::cerr << "Failed to free device vector f (error code " << hipGetErrorString(err) << ")!" << std::endl;
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_c);
    if (err != hipSuccess)
    {
        std::cerr << "Failed to free device vector c (error code " << hipGetErrorString(err) << ")!" << std::endl;
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_g);
    if (err != hipSuccess)
    {
        std::cerr << "Failed to free device vector g (error code " << hipGetErrorString(err) << ")!" << std::endl;
        exit(EXIT_FAILURE);
    }
    
    err = hipFree(d_g_m);
    if (err != hipSuccess)
    {
        std::cerr << "Failed to free device vector g_m (error code " << hipGetErrorString(err) << ")!" << std::endl;
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_tmp1);
    if (err != hipSuccess)
    {
        std::cerr << "Failed to free device vector tmp1 (error code " << hipGetErrorString(err) << ")!" << std::endl;
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_tmp2);
    if (err != hipSuccess)
    {
        std::cerr << "Failed to free device vector tmp2 (error code " << hipGetErrorString(err) << ")!" << std::endl;
        exit(EXIT_FAILURE);
    }

    Image output = ptr2image(output_ptr, width, height);
    gputime_gpu.stop();

    std::cout << "Total time Elapsed - GPU: " << gputime_gpu.elapsed_time() << " ms" << std::endl;

    std::cout << "BaselinePSNR: " << psnr(ptr2image(image_ptr, width, height), target_image) << std::endl;
    std::cout << "PSNR: " << psnr(output, target_image) << std::endl;

    saveImage(output, output_file);
    std::cout << "Image saved to: " << output_file << std::endl;
}


double *image2ptr(const Image& input)
{
    int width  = input[0][0].size();
    int height = input[0].size();

    double *ptr = new (std::nothrow) double[3*height*width];
    int idx = 0;
    for (int i = 0; i < height; ++i)
    {
        for (int j =0; j < width; ++j)
        {
            ptr[idx++] = input[0][i][j];
            ptr[idx++] = input[1][i][j];
            ptr[idx++] = input[2][i][j];
        }
    }

    return ptr;
}


double *matrix2ptr(const Matrix &input)
{
    int width  = input[0].size();
    int height = input.size();

    double *ptr = new (std::nothrow) double[height*width];
    int idx = 0;
    for (int i = 0; i < height; ++i)
        for (int j =0; j < width; ++j)
            ptr[idx++] = input[i][j];

    return ptr;
}


Image ptr2image(const double *input, const int width, const int height)
{
    Image output(3, createMatrix(height, width) );

    int idx = 0;
    for (int i = 0; i < height; ++i)
    {
        for (int j = 0 ; j < width; ++j)
        {
            output[0][i][j] = input[idx++];
            output[1][i][j] = input[idx++];
            output[2][i][j] = input[idx++];
        }
    }
    return output;
}


Matrix gaussian(const int height, const int width, const double sigma)
{
    Matrix kernel = createMatrix(height, width);
    double sum = 0.0;

    for (int i = 0; i < height; i++)
    {
        for (int j = 0; j < width; j++)
        {
            kernel[i][j] = exp(-(i * i + j * j) / (2 * sigma * sigma)) / (2 * M_PI * sigma * sigma);
            sum += kernel[i][j];
        }
    }

    for (int i = 0; i < height; i++)
        for (int j = 0; j < width; j++)
            kernel[i][j] /= sum;
            
    return kernel;
}

Matrix sharpen(const int height, const int width)
{
    Matrix kernel = createMatrix(height, width);

    kernel[0][0] = 0;
    kernel[1][0] = -1;
    kernel[2][0] = 0;
    kernel[1][0] = -1;
    kernel[1][1] = 4;
    kernel[1][2] = -1;
    kernel[2][0] = 0;
    kernel[2][1] = -1;
    kernel[2][2] = 0;

    return kernel;
}

Matrix createMatrix(const int height, const int width)
{
    return Matrix(height, Array(width, 0));
}
