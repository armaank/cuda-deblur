
#include <hip/hip_runtime.h>
   

/////////////////////////////////////////////////////////////////////////////////
/////////////////////////////// Kernel Functions ////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////

__global__
void kernel_internalMemcpy(float *dest,const float *from, const uint W, const uint H)
{
    const int start_idx = blockDim.x * blockIdx.x + threadIdx.x;
    const int stride    = blockDim.x * gridDim.x;
    const int max_val   = H*W;
    
    for (int idx = start_idx; idx < max_val; idx += stride) 
        dest[idx] = from[idx];
}


/**
 * KernelConvolve - Computes the discrete convolution C=A*B. 
 * The dimensions of A, B, and C are all assumed to be W x H.
 */
__global__
void kernel_convolve(const float *A, const float *B, float *C, const uint W, const uint H)
{
    const int start_idx = blockDim.x * blockIdx.x + threadIdx.x;
    const int stride    = blockDim.x * gridDim.x;
    const int max_val   = H*W;

    // will not execute if start_idx<max_val
    for (int c_idx=start_idx; c_idx<max_val; c_idx += stride) 
    {
        C[c_idx] = 0;

        // get the single c_idx term in 2D terms
        int i = c_idx % W;
        int j = c_idx - W*i;

        for (int m=0; m<H; ++m)
        {
            for (int n=0; j<W; ++n)
            {
                int cur_idx = m*W + n;
                C[c_idx] += A[cur_idx] * B[ (i-m)*W+(j-n) ];
            }
        }
    }
}
/* convolve a 2D image (RBG) w/ a filter */
__global__



/** 
 * KernelElementWiseDivision - Executes an elementwise division C = A/B.
 * The dimensions of A, B, and C are all assumed to be W x H.
 */
__global__
void kernel_elementWiseDivision(const float *A, const float *B, float *C, const uint W, const uint H)
{
    const int start_idx = blockDim.x * blockIdx.x + threadIdx.x;
    const int stride    = blockDim.x * gridDim.x;
    const int max_val   = 3*H*W;

    // will not execute if start_idx<max_val
    for (int c_idx=start_idx; c_idx<max_val; c_idx += stride) 
    {
        C[c_idx] = B[c_idx]==0 ? 999999999 : A[c_idx]/B[c_idx];
    }
}

/** 
 * KernelElementWiseMultiplication - Executes an elementwise multiplication C = A*B.
 * The dimensions of A, B, and C are all assumed to be W x H.
 */
__global__
void kernel_elementWiseMultiplication(const float *A, const float *B, float *C, const uint W, const uint H)
{
    const int start_idx = blockDim.x * blockIdx.x + threadIdx.x;
    const int stride    = blockDim.x * gridDim.x;
    const int max_val   = 3*H*W;

    // will not execute if start_idx > max_val
    for(int c_idx=start_idx; c_idx<max_val; c_idx += stride) 
    {
        C[c_idx] = A[c_idx]*B[c_idx];
    }
}


/////////////////////////////////////////////////////////////////////////////////
///////////////////////////////// CPU Functions ////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////

/**
  * updateUnderlyingImg - Executes a Lucy-Richardson iteration to get an updated
  * underlying image (updates the values of f).
  *   c - The blurred image.
  *   g - The PSF.
  *   f - The underlying image we are trying to recover.
  *   H - Height of the image.
  *   W - Width of the image.
  */
  void updateUnderlyingImg(const float *c, const float *g, const float *g_m, float *f, float *tmp1, float *tmp2, const uint W, const uint H)
  {
      hipError_t err = hipSuccess; // Error code to check return values for CUDA calls

      int threadsPerBlock = 256;
      int blocksPerGrid =(H*W + threadsPerBlock - 1) / threadsPerBlock;
  
      kernel_convolve<<<blocksPerGrid, threadsPerBlock>>>(f, g, tmp1, W, H);
      err = hipGetLastError();
      if (err != hipSuccess)
      {
          fprintf(stderr, "Failed to launch KernelConvolve kernel (error code %s)!\n", hipGetErrorString(err));
          exit(EXIT_FAILURE);
      }
  
      kernel_elementWiseDivision<<<blocksPerGrid, threadsPerBlock>>>(c, tmp1, tmp2, W, H);
      err = hipGetLastError();
      if (err != hipSuccess)
      {
          fprintf(stderr, "Failed to launch KernelElementWiseDivision kernel (error code %s)!\n", hipGetErrorString(err));
          exit(EXIT_FAILURE);
      }
  
      kernel_convolve<<<blocksPerGrid, threadsPerBlock>>>(tmp2, g_m, tmp1, W, H);
      err = hipGetLastError();
      if (err != hipSuccess)
      {
          fprintf(stderr, "Failed to launch KernelConvolve kernel (error code %s)!\n", hipGetErrorString(err));
          exit(EXIT_FAILURE);
      }
  
      kernel_elementWiseMultiplication<<<blocksPerGrid, threadsPerBlock>>>(tmp1, f, tmp2, W, H);
      err = hipGetLastError();
      if (err != hipSuccess)
      {
          fprintf(stderr, "Failed to launch KernelElementWiseMultiplication kernel (error code %s)!\n", hipGetErrorString(err));
          exit(EXIT_FAILURE);
      }
  
      kernel_internalMemcpy<<<blocksPerGrid, threadsPerBlock>>>(f, tmp2, W, H);
      err = hipGetLastError();
      if (err != hipSuccess)
      {
          fprintf(stderr, "Failed to launch KernelInternalMemcpy kernel (error code %s)!\n", hipGetErrorString(err));
          exit(EXIT_FAILURE);
      }
  }